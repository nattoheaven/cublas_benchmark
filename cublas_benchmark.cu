/*
 * Copyright (C) 2012  NISHIMURA Ryohei
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>

#define CUDA_SAFE_CALL(E) do {                                          \
    hipError_t e = (E);                                                \
    if (e != hipSuccess) {                                             \
      printf("line %d: CUDA error: %s\n", __LINE__, hipGetErrorString(e)); \
      exit(-2);                                                         \
    }                                                                   \
  } while (false)

#define CUBLAS_SAFE_CALL(S) do {                                \
    hipblasStatus_t s = (S);                                     \
    if (s != HIPBLAS_STATUS_SUCCESS) {                           \
      printf("line %d: CUBLAS error: %d\n", __LINE__, s);       \
      exit(-3);                                                 \
    }                                                           \
  } while (false)

#define CHECKZERO(X) do {                               \
    if ((X) <= 0) {                                     \
      printf("error: " #X " = %d <= 0!\n", (X));        \
      exit(-1);                                         \
    }                                                   \
  } while(false)

#define CHECKMINMAX(X, Y) do {                                  \
    if ((X) > (Y)) {                                            \
      printf("error: " #X " = %d > %d = " #Y "!\n", (X), (Y));  \
      exit(-1);                                                 \
    }                                                           \
  } while(false)

template<typename T> void
gemm(hipblasHandle_t handle, int m, int n, int k,
     const T *a, int lda, const T *b, int ldb,
     T *c, int ldc);

template<> void
gemm<float>(hipblasHandle_t handle, int m, int n, int k,
            const float *a, int lda,
            const float *b, int ldb,
            float *c, int ldc)
{
  float alpha = 1.0f;
  float beta = 1.0f;
  CUBLAS_SAFE_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               m, n, k,
                               &alpha, a, lda, b, ldb, &beta, c, ldc));
}

template<> void
gemm<double>(hipblasHandle_t handle, int m, int n, int k,
             const double *a, int lda,
             const double *b, int ldb,
             double *c, int ldc)
{
  double alpha = 1.0;
  double beta = 1.0;
  CUBLAS_SAFE_CALL(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               m, n, k,
                               &alpha, a, lda, b, ldb, &beta, c, ldc));
}

template<> void
gemm<hipComplex>(hipblasHandle_t handle, int m, int n, int k,
                const hipComplex *a, int lda,
                const hipComplex *b, int ldb,
                hipComplex *c, int ldc)
{
  hipComplex alpha = make_hipComplex(1.0f, 0.0f);
  hipComplex beta = make_hipComplex(1.0f, 0.0f);
  CUBLAS_SAFE_CALL(hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               m, n, k,
                               &alpha, a, lda, b, ldb, &beta, c, ldc));
}

template<> void
gemm<hipDoubleComplex>(hipblasHandle_t handle, int m, int n, int k,
                      const hipDoubleComplex *a, int lda,
                      const hipDoubleComplex *b, int ldb,
                      hipDoubleComplex *c, int ldc)
{
  hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(1.0, 0.0);
  CUBLAS_SAFE_CALL(hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               m, n, k,
                               &alpha, a, lda, b, ldb, &beta, c, ldc));
}

template<typename T> double
calc_gflops(int m, int n, int k, float ms);

template<> double
calc_gflops<float>(int m, int n, int k, float ms)
{
  return 2.0 * m * n * k / ms * 1.0e-6;
}

template<> double
calc_gflops<double>(int m, int n, int k, float ms)
{
  return 2.0 * m * n * k / ms * 1.0e-6;
}

template<> double
calc_gflops<hipComplex>(int m, int n, int k, float ms)
{
  return 8.0 * m * n * k / ms * 1.0e-6;
}

template<> double
calc_gflops<hipDoubleComplex>(int m, int n, int k, float ms)
{
  return 8.0 * m * n * k / ms * 1.0e-6;
}

template<typename T> void
benchmark(int m, int n, int k,
          int ldamin, int ldamax, int ldastep,
          int ldbmin, int ldbmax, int ldbstep,
          int ldcmin, int ldcmax, int ldcstep)
{
  hipblasHandle_t handle;
  CUBLAS_SAFE_CALL(hipblasCreate(&handle));
  T *a, *b, *c;
  CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void **>(&a),
                            k * ldamax * sizeof(T)));
  CUDA_SAFE_CALL(hipMemset(a, 0, k * ldamax * sizeof(T)));
  CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void **>(&b),
                            n * ldbmax * sizeof(T)));
  CUDA_SAFE_CALL(hipMemset(b, 0, n * ldbmax * sizeof(T)));
  CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void **>(&c),
                            n * ldcmax * sizeof(T)));
  CUDA_SAFE_CALL(hipMemset(c, 0, n * ldcmax * sizeof(T)));
  hipStream_t stream;
  CUDA_SAFE_CALL(hipStreamCreate(&stream));
  CUBLAS_SAFE_CALL(hipblasSetStream(handle, stream));
  hipEvent_t start, end;
  CUDA_SAFE_CALL(hipEventCreate(&start));
  CUDA_SAFE_CALL(hipEventCreate(&end));
  printf("M: %d\tN: %d\tK: %d\n", m, n, k);
  printf("LDA\tLDB\tLDC\tSecond\tGFLOPS\n");
  for (int ldc = ldcmin; ldc <= ldcmax; ldc += ldcstep) {
    for (int ldb = ldbmin; ldb <= ldbmax; ldb += ldbstep) {
      for (int lda = ldamin; lda <= ldamax; lda += ldastep) {
        CUDA_SAFE_CALL(hipEventRecord(start, stream));
        gemm(handle, m, n, k, a, lda, b, ldb, c, ldc);
        CUDA_SAFE_CALL(hipEventRecord(end, stream));
        CUDA_SAFE_CALL(hipEventSynchronize(end));
        float ms;
        CUDA_SAFE_CALL(hipEventElapsedTime(&ms, start, end));
        double gflops = calc_gflops<T>(m, n, k, ms);
        printf("%d\t%d\t%d\t%.6f\t%.6f\n", lda, ldb, ldc, ms, gflops);
      }
    }
  }
  CUDA_SAFE_CALL(hipEventDestroy(end));
  CUDA_SAFE_CALL(hipEventDestroy(start));
  CUDA_SAFE_CALL(hipStreamDestroy(stream));
  CUBLAS_SAFE_CALL(hipblasDestroy(handle));
}

int
main(int argc, char **argv)
{
  if (argc < 14) {
    printf("usage: %s typename([sdcz]) m n k ldamin ldamax ldastep ldbmin ldbmax ldbstep ldcmin ldcmax ldcstep\n", argv[0]);
    return 0;
  }
  ptrdiff_t iarg = 0;
  char *type = argv[++iarg];
  int m = atoi(argv[++iarg]);
  int n = atoi(argv[++iarg]);
  int k = atoi(argv[++iarg]);
  int ldamin = atoi(argv[++iarg]);
  int ldamax = atoi(argv[++iarg]);
  int ldastep = atoi(argv[++iarg]);
  int ldbmin = atoi(argv[++iarg]);
  int ldbmax = atoi(argv[++iarg]);
  int ldbstep = atoi(argv[++iarg]);
  int ldcmin = atoi(argv[++iarg]);
  int ldcmax = atoi(argv[++iarg]);
  int ldcstep = atoi(argv[++iarg]);
  CHECKZERO(m);
  CHECKZERO(n);
  CHECKZERO(k);
  CHECKZERO(ldamin);
  CHECKMINMAX(m, ldamin);
  CHECKMINMAX(ldamin, ldamax);
  CHECKZERO(ldastep);
  CHECKZERO(ldbmin);
  CHECKMINMAX(k, ldbmin);
  CHECKMINMAX(ldbmin, ldbmax);
  CHECKZERO(ldbstep);
  CHECKZERO(ldcmin);
  CHECKMINMAX(m, ldcmin);
  CHECKMINMAX(ldcmin, ldcmax);
  CHECKZERO(ldcstep);

  switch (type[0]) {
  case 'S':
  case 's':
    benchmark<float>(m, n, k,
                     ldamin, ldamax, ldastep,
                     ldbmin, ldbmax, ldbstep,
                     ldcmin, ldcmax, ldcstep);
    break;
  case 'D':
  case 'd':
    benchmark<double>(m, n, k,
                      ldamin, ldamax, ldastep,
                      ldbmin, ldbmax, ldbstep,
                      ldcmin, ldcmax, ldcstep);
    break;
  case 'C':
  case 'c':
    benchmark<hipComplex>(m, n, k,
                         ldamin, ldamax, ldastep,
                         ldbmin, ldbmax, ldbstep,
                         ldcmin, ldcmax, ldcstep);
    break;
  case 'Z':
  case 'z':
    benchmark<hipDoubleComplex>(m, n, k,
                               ldamin, ldamax, ldastep,
                               ldbmin, ldbmax, ldbstep,
                               ldcmin, ldcmax, ldcstep);
    break;
  default:
    printf("invaild typename: %s\n", type);
    return -1;
  }

  return 0;
}
